#include "hip/hip_runtime.h"
//#include <stdio.h>
//#include <stdint.h>
//#include <stdlib.h>
//#include <fstream>
#include "mnist.h"
#include <string>
#include <fstream>
#include <array>
#include <stdio.h>
#include <iostream>

#define DEBUG 2


using namespace std;

__global__ void getback(uint8_t *device_in, uint8_t *device_out, int NUM_IMGS, int NUM_PIXELS) {  
  int idx = NUM_PIXELS*(blockIdx.x*blockDim.x + threadIdx.x);
  if(idx < NUM_IMGS * NUM_PIXELS){
    for(int i=0; i< NUM_PIXELS;i++){
      device_out[idx+i] = device_in[idx+i];
    }
  }
}

int main(int argc, char **argv){
  // dataset paths
  string base_dir = "data/";
  string img_path = base_dir+"train-images-idx3-ubyte";
  string label_path = base_dir+"train-labels-idx1-ubyte";

  // 60000/1024 ~ 59, where 1024 is MAX_THREADS
  const int NUM_THREADS = 1024;
  const int NUM_BLOCKS = 59;

  // dataset parameters
  uint32_t *NUM_IMGS = (uint32_t*) malloc(sizeof(uint32_t));
  uint32_t *NUM_LABELS = (uint32_t*) malloc(sizeof(uint32_t));
  uint32_t *NUM_COLS = (uint32_t*) malloc(sizeof(uint32_t));
  uint32_t *NUM_ROWS = (uint32_t*) malloc(sizeof(uint32_t));
  uint32_t *NUM_PIXELS = (uint32_t*) malloc(sizeof(uint32_t));
  uint32_t *NUM_BYTES = (uint32_t*) malloc(sizeof(uint32_t));

  // load host_in with the dataset
  uint8_t *host_in = load_mnist(img_path.c_str(), label_path.c_str(), NUM_IMGS, NUM_LABELS, NUM_COLS, 
             NUM_ROWS, NUM_PIXELS, NUM_BYTES);

  uint8_t *host_out = (uint8_t*) malloc(*NUM_BYTES); 

  uint8_t *device_in;
  uint8_t *device_out;

  // allocate device memory
  hipMalloc(&device_in, *NUM_BYTES);
  hipMalloc(&device_out, *NUM_BYTES);

  // copy host_in into device_in
  hipMemcpy(device_in, host_in, *NUM_BYTES, hipMemcpyHostToDevice);
  //hipMemcpy(device_out, device_in, *NUM_BYTES, hipMemcpyDeviceToDevice); // copy pointers bypassing global function
  getback<<<NUM_BLOCKS,NUM_THREADS>>>(device_in, device_out, *NUM_IMGS, *NUM_PIXELS);
  //hipMemcpy(host_out, device_in, *NUM_BYTES, hipMemcpyDeviceToHost); // copy pointers bypassing device_out 
  hipMemcpy(host_out, device_out, *NUM_BYTES, hipMemcpyDeviceToHost);

  // free device memory
  hipFree(device_in);
  hipFree(device_out);
  
  // free host memory
  hipHostFree(host_in);
  hipHostFree(host_out);

#if DEBUG==2
  // render image data in terminal!
  string color;
  int bin_offset = 232; // greyscale color range is 232-255
  string print_exp;
  for(int j=0;j<*NUM_IMGS * *NUM_PIXELS;j++){
    if(j%28==0){
	    if(j/28==0){printf("\n");}
	      printf("\n");
        printf("%3d: ",int(j/28));
    }

    color = to_string(host_out[j]/24+(host_out[j]%24!=0)+bin_offset); // 255-232+1 = 24 bins (some aliasing)
    print_exp = "\u001b[48;5;"+color+"m  "; // 8 bit 256 color code
    printf("%s",print_exp.c_str());
  }
  cout<<endl;
#endif
}