//#include <stdio.h>
//#include <stdint.h>
//#include <stdlib.h>
//#include <fstream>


#include <hip/hip_runtime.h>
#include <string>
#include <fstream>
#include <array>
#include <stdio.h>
#include <iostream>

#define DEBUG 2


using namespace std;

__global__ void getback(uint8_t *device_in, uint8_t *device_out, int NUM_COLS, int NUM_ROWS) {  
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if(idx < NUM_COLS*NUM_ROWS){
    device_out[idx] = device_in[idx];
  }
}

int main(){
  const int NUM_THREADS = 1024;
  const int NUM_BLOCKS = 45938;
  const int NUM_COLS = 60000;
  const int NUM_ROWS = 784;
  const int NUM_BYTES = NUM_COLS * NUM_ROWS * sizeof(uint8_t);

  uint8_t *host_in = (uint8_t*) malloc(NUM_BYTES);
  uint8_t *host_out = (uint8_t*) malloc(NUM_BYTES); 

  uint8_t *device_in;
  uint8_t *device_out;

  hipMalloc(&device_in, NUM_BYTES);
  hipMalloc(&device_out, NUM_BYTES);

  int count=0;
    for(int i=0; i< NUM_COLS*NUM_ROWS; i++){
      *(host_in + i) = count;
      if(count == 255){count = 0;}
      else{count++;}
    }
    printf("\n");
  
#if DEBUG==1

  for(int i=0; i < NUM_COLS*NUM_ROWS; i++){
    printf("%d",host_in[i]);
  }
  printf("\n\n");
#endif

  hipMemcpy(device_in, host_in, NUM_BYTES, hipMemcpyHostToDevice);
//  cudaMemcpy(device_out, device_in, NUM_BYTES, cudaMemcpyDeviceToDevice); // copy pointers bypassing global function
  getback<<<NUM_BLOCKS,NUM_THREADS>>>(device_in, device_out, NUM_COLS, NUM_ROWS);

  hipMemcpy(host_out, device_out, NUM_BYTES, hipMemcpyDeviceToHost);

  hipFree(device_in);
  hipFree(device_out);
  
  hipHostFree(host_in);
  hipHostFree(host_out);

#if DEBUG==2
  for(int i=0; i<NUM_COLS*NUM_ROWS; i++){
    printf("%d ", *(host_out + i));
  }
  printf("\n");
#endif
}