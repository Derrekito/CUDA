//#include <stdio.h>
//#include <stdint.h>
//#include <stdlib.h>
//#include <fstream>


#include <hip/hip_runtime.h>
#include <string>
#include <fstream>
#include <array>
#include <stdio.h>
#include <iostream>


#define DEBUG 2


using namespace std;

__global__ void getback(uint8_t *device_in, uint8_t *device_out, int NUM_ELM) {  
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if(idx < NUM_ELM){
    device_out[idx] = device_in[idx];
  }
}

int main(){
  const int NUM_THREADS = 1024;
  const int NUM_BLOCKS = 2;
  const int NUM_ELM = 2048;
  const int NUM_BYTES = NUM_ELM * sizeof(uint8_t);

  uint8_t *host_in = (uint8_t*) malloc(NUM_BYTES);
  uint8_t *host_out = (uint8_t*) malloc(NUM_BYTES);

  uint8_t *device_in;
  uint8_t *device_out;

  hipMalloc(&device_in, NUM_BYTES);
  hipMalloc(&device_out, NUM_BYTES);

  int count=0;
  for(int i=0; i<NUM_ELM; i++){
    *(host_in + i) = count;
    if(count == 255){count = 0;}
    else{count++;}
  }
#if DEBUG==1
  for(int i=0; i< NUM_ELM; i++){
    printf("%d ", *(host_in + i));
  }
  printf("\n\n");
#endif

  hipMemcpy(device_in, host_in, NUM_BYTES, hipMemcpyHostToDevice);
//  cudaMemcpy(device_out, device_in, NUM_BYTES, cudaMemcpyDeviceToDevice); // copy pointers bypassing global function
  getback<<<NUM_BLOCKS,NUM_THREADS>>>(device_in, device_out, NUM_ELM);

  hipMemcpy(host_out, device_out, NUM_BYTES, hipMemcpyDeviceToHost);

  hipFree(device_in);
  hipFree(device_out);
#if DEBUG==2
  for(int i=0; i<NUM_ELM; i++){
    printf("%d ", *(host_out + i));
  }
  printf("\n");
#endif
}