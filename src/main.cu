#include "hip/hip_runtime.h"
//#include <stdio.h>
//#include <stdint.h>
//#include <stdlib.h>
//#include <fstream>
#include "mnist.h"
#include <string>
#include <fstream>
#include <array>
#include <stdio.h>
#include <iostream>

#define DEBUG 2


using namespace std;

__global__ void getback(uint8_t *device_in, uint8_t *device_out, int NUM_IMGS, int NUM_PIXELS) {  
  /*
  int idx = 784*(blockIdx.x*blockDim.x + threadIdx.x);
  if(idx < NUM_IMGS){
    for(int i=0; i< NUM_PIXELS;i++){
      device_out[idx+i] = 255;//device_in[idx+i];
    }
  }*/
  for(int i=0;i<NUM_IMGS * NUM_PIXELS; i++){
    device_out[i] = device_in[i];
  }
}

int main(int argc, char **argv){
  string base_dir = "data/";
  string img_path = base_dir+"train-images-idx3-ubyte";
  string label_path = base_dir+"train-labels-idx1-ubyte";

//  const int NUM_THREADS = 1024;
//  const int NUM_BLOCKS = 59;
const int NUM_THREADS = 1;
const int NUM_BLOCKS = 1;
  /*
  const int NUM_IMGS = 60000;
  const int NUM_COLS = 28;
  const int NUM_ROWS = 28;
  const int NUM_PIXELS = NUM_COLS * NUM_ROWS;
  const int NUM_BYTES = NUM_IMGS * NUM_PIXELS * sizeof(uint8_t);
*/
  uint32_t *NUM_IMGS = (uint32_t*) malloc(sizeof(uint32_t));
  uint32_t *NUM_LABELS = (uint32_t*) malloc(sizeof(uint32_t));
  uint32_t *NUM_COLS = (uint32_t*) malloc(sizeof(uint32_t));
  uint32_t *NUM_ROWS = (uint32_t*) malloc(sizeof(uint32_t));
  uint32_t *NUM_PIXELS = (uint32_t*) malloc(sizeof(uint32_t));
  uint32_t *NUM_BYTES = (uint32_t*) malloc(sizeof(uint32_t));

  uint8_t *host_in = (uint8_t*) malloc(60000*28*28*sizeof(uint8_t));

  load_mnist(img_path.c_str(), label_path.c_str(), NUM_IMGS, NUM_LABELS, NUM_COLS, 
             NUM_ROWS, NUM_PIXELS, NUM_BYTES, host_in);

  uint8_t *host_out = (uint8_t*) malloc(*NUM_BYTES); 

  uint8_t *device_in;
  uint8_t *device_out;

  hipMalloc(&device_in, *NUM_BYTES);
  hipMalloc(&device_out, *NUM_BYTES);

  int count=0;
  for(int i=0; i< *NUM_COLS * *NUM_ROWS; i++){
    *(host_in + i) = count;
    if(count == 255){count = 0;}
    else{count++;}
  }

  hipMemcpy(device_in, host_in, *NUM_BYTES, hipMemcpyHostToDevice);
  //hipMemcpy(device_out, device_in, *NUM_BYTES, hipMemcpyDeviceToDevice); // copy pointers bypassing global function
  //getback<<<NUM_BLOCKS,NUM_THREADS>>>(device_in, device_out, *NUM_COLS, *NUM_ROWS);
  //hipMemcpy(host_out, device_in, *NUM_BYTES, hipMemcpyDeviceToHost); // copy pointers bypassing device_out 
  hipMemcpy(host_out, device_out, *NUM_BYTES, hipMemcpyDeviceToHost);

  hipFree(device_in);
  hipFree(device_out);
  
  hipHostFree(host_in);
  hipHostFree(host_out);

#if DEBUG==2
  string color;
  int bin_offset = 232; // greyscale color range is 232-255
  string print_exp;
  for(int j=0;j<*NUM_IMGS * *NUM_ROWS * *NUM_COLS;j++){
    if(j%28==0){
	    if(j/28==0){printf("\n");}
	      printf("\n");
        printf("%3d: ",int(j/28));
    }

    color = to_string(host_out[j]/24+(host_out[j]%24!=0)+bin_offset); // 255-232+1 = 24 bins
    //printf("%s ",color.c_str());
    print_exp = "\u001b[48;5;"+color+"m  "; // 8 bit 256 color code
    printf("%s",print_exp.c_str());
  }
  cout<<endl;
#endif
}